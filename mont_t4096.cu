// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#ifndef __CUDA_ARCH__
#define __CUDA_ARCH__
#endif
#if defined(__CUDA_ARCH__) && !defined(__SPPARK_FF_MONT_T_CUH__)
# define __SPPARK_FF_MONT_T_CUH__

# include <cstddef>
# include <cstdint>

# define inline __device__ __forceinline__
# ifdef __GNUC__
#  define asm __asm__ __volatile__
# else
#  define asm asm volatile
# endif

// 0x0e0a77c19a07df2f666ea36f7879462c0a78eb28f5c70b3dd35d438dc58f0d9d
// 0x0e0a77c19a07df2f666ea36f7879462c0a78eb28f5c70b3dd35d438dc58f0d9d
namespace device {
#define TO_CUDA_T(limb64) (uint32_t)(limb64), (uint32_t)(limb64>>32)
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_P[8] = {
        TO_CUDA_T(0x3c208c16d87cfd47), TO_CUDA_T(0x97816a916871ca8d),
        TO_CUDA_T(0xb85045b68181585d), TO_CUDA_T(0x30644e72e131a029)
    };
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_RR[8] = { /* (1<<512)%P */
        TO_CUDA_T(0xf32cfc5b538afa89), TO_CUDA_T(0xb5e71911d44501fb),
        TO_CUDA_T(0x47ab1eff0a417ff6), TO_CUDA_T(0x06d89f71cab8351f),
    };
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_one[8] = { /* (1<<256)%P */
        TO_CUDA_T(0xd35d438dc58f0d9d), TO_CUDA_T(0x0a78eb28f5c70b3d),
        TO_CUDA_T(0x666ea36f7879462c), TO_CUDA_T(0x0e0a77c19a07df2f)
    };
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_Px4[8] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xf082305b61f3f51c), TO_CUDA_T(0x5e05aa45a1c72a34),
        TO_CUDA_T(0xe14116da06056176), TO_CUDA_T(0xc19139cb84c680a6)
    };
    //n`(rr_1-nn`=1) 0xd76b780d00bad4acbbc3ac183dcfc3f073dd0cc126dc4ece43b5e10de0f8076
    static __device__ __constant__ const uint32_t ALT_BN128_M0 = 0xe4866389;


    static __device__ __constant__ __align__(16) const uint32_t MNT4753_Fr_P[24] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xd90776e240000001), TO_CUDA_T(0x4ea099170fa13a4f),
        TO_CUDA_T(0xd6c381bc3f005797), TO_CUDA_T(0xb9dff97634993aa4),
        TO_CUDA_T(0x3eebca9429212636), TO_CUDA_T(0xb26c5c28c859a99b),
        TO_CUDA_T(0x99d124d9a15af79d), TO_CUDA_T(0x07fdb925e8a0ed8d),
        TO_CUDA_T(0x5eb7e8f96c97d873), TO_CUDA_T(0xb7f997505b8fafed),
        TO_CUDA_T(0x10229022eee2cdad), TO_CUDA_T(0x0001c4c62d92c411)
    };

    
    static __device__ __constant__ const uint32_t MNT4753_Fr_M0 = 0x3fffffff;


    static __device__ __constant__ __align__(16) const uint32_t BLS12_377_P[12] = {
        TO_CUDA_T(0x8508c00000000001), TO_CUDA_T(0x170b5d4430000000),
        TO_CUDA_T(0x1ef3622fba094800), TO_CUDA_T(0x1a22d9f300f5138f),
        TO_CUDA_T(0xc63b05c06ca1493b), TO_CUDA_T(0x01ae3a4617c510ea)
    };

    
    static __device__ __constant__ /*const*/ uint32_t BLS12_377_M0 = 0xffffffff;

         static __device__ __constant__ __align__(16) const uint32_t curve448_P[14] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xfffffffeffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff)
    };


    static __device__ __constant__ /*const*/ uint32_t curve448_M0 = 0x1;


 static __device__ __constant__ __align__(16) const uint32_t Curve1024_P0[32] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xfffffffffffffffe), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff)
    };



    static __device__ __constant__ /*const*/ uint32_t Curve1024_M0 = 0x1;

 static __device__ __constant__ __align__(16) const uint32_t Curve2048_P0[64] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xfffffffffffffffe), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff)
    };



    static __device__ __constant__ /*const*/ uint32_t Curve2048_M0 = 0x1;

    static __device__ __constant__ __align__(16) const uint32_t Curve4096_P0[128] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xfffffffffffffffe), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff)
    };



    static __device__ __constant__ /*const*/ uint32_t Curve4096_M0 = 0x1;
}

void __global__  func1(uint32_t res[]);
//
// To instantiate declare modulus as __device__ __constant___ const and
// complement it with its factual bit-length and the corresponding 32-bit
// Motgomery factor. Bit-length has to be such that (N+31)/32 is even
// and not less than 4.
//
// Special note about M0 being declared as uint32_t& [as opposed to just
// uint32_t]. It was noted that if M0 is 0xffffffff, CUDA compiler
// generates suboptimal code for Montgomery reduction. The way to work
// around the problem is to prevent compiler from viewing it as constant.
// For this reason it's suggested to declare the parameter as following:
//
//    __device__ __constant__ /*const*/ my_M0 = <literal>;
//
template<const size_t N, const uint32_t MOD[(N+31)/32], const uint32_t& M0,
         const uint32_t RR[(N+31)/32], const uint32_t ONE[(N+31)/32],
         const uint32_t MODx[(N+31)/32] = MOD>
class __align__(((N+63)/64)&1 ? 8 : 16) mont_t {
        
    public:
        static const size_t nbits = N;
        static constexpr size_t __device__ bit_length() { return N; }
        static const uint32_t degree = 1;
        using mem_t = mont_t;
        static const size_t n = (N+31)/32;
        
    public:
        uint32_t even[n];

    private:
        static inline void mul_n(uint32_t* acc, const uint32_t* a, uint32_t bi,
        size_t n_=n)
        {
            for (size_t j = 0; j < n_; j += 2)
                asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
                    : "=r"(acc[j]), "=r"(acc[j+1])
                    : "r"(a[j]), "r"(bi));
        }

        static inline void cmad_n(uint32_t* acc, const uint32_t* a, uint32_t bi,
                                size_t n_=n)
        {
            asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
                : "+r"(acc[0]), "+r"(acc[1])
                : "r"(a[0]), "r"(bi));
            for (size_t j = 2; j < n_; j += 2)
                asm("madc.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
                    : "+r"(acc[j]), "+r"(acc[j+1])
                    : "r"(a[j]), "r"(bi));
            // return carry flag
        }

        static inline void cadd_n(uint32_t* acc, const uint32_t* a, size_t n_=n)
        {
            asm("add.cc.u32 %0, %0, %1;" : "+r"(acc[0]) : "r"(a[0]));
            for (size_t i = 1; i < n_; i++)
                asm("addc.cc.u32 %0, %0, %1;" : "+r"(acc[i]) : "r"(a[i]));
            // return carry flag
        }

        class wide_t {
        private:
            union {
                uint32_t even[2*n];
                mont_t s[2];
            };

        public:
            inline uint32_t& operator[](size_t i)               { return even[i]; }
            inline const uint32_t& operator[](size_t i) const   { return even[i]; }
            inline operator mont_t()
            {
                s[0].mul_by_1();
                return s[0] + s[1];
            }
            inline void final_sub(uint32_t carry, uint32_t* tmp)
            {   s[1].final_sub(carry, tmp);   }

            inline wide_t() {}

        private:
            static inline void mad_row(uint32_t* odd, uint32_t* even,
                                    const uint32_t* a, uint32_t bi, size_t n_=n)
            {
                cmad_n(odd, a+1, bi, n_-2);
                asm("madc.lo.cc.u32 %0, %2, %3, 0; madc.hi.u32 %1, %2, %3, 0;"
                    : "=r"(odd[n_-2]), "=r"(odd[n_-1])
                    : "r"(a[n_-1]), "r"(bi));

                cmad_n(even, a, bi, n_);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n_-1]));
            }

        public:
            inline wide_t(const mont_t& a, const mont_t& b)     //// |a|*|b|
            {
                size_t i = 0;
                uint32_t odd[2*n-2];

                mul_n(even, &a[0], b[0]);
                mul_n(odd,  &a[1], b[0]);
                ++i; mad_row(&even[i+1], &odd[i-1], &a[0], b[i]);

                #pragma unroll
                while (i < n-2) {
                    ++i; mad_row(&odd[i],    &even[i],  &a[0], b[i]);
                    ++i; mad_row(&even[i+1], &odd[i-1], &a[0], b[i]);
                }

                // merge |even| and |odd|
                cadd_n(&even[1], &odd[0], 2*n-2);
                asm("addc.u32 %0, %0, 0;" : "+r"(even[2*n-1]));
            }
        private:
            static inline void qad_row(uint32_t* odd, uint32_t* even,
                                    const uint32_t* a, uint32_t bi, size_t n)
            {
                cmad_n(odd, a, bi, n-2);
                asm("madc.lo.cc.u32 %0, %2, %3, 0; madc.hi.u32 %1, %2, %3, 0;"
                    : "=r"(odd[n-2]), "=r"(odd[n-1])
                    : "r"(a[n-2]), "r"(bi));

                cmad_n(even, a+1, bi, n-2);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            }
        public:
            inline wide_t(const mont_t& a)                      //// |a|**2
            {
                size_t i = 0, j;
                uint32_t odd[2*n-2];

                // perform |a[i]|*|a[j]| for all j>i
                mul_n(even+2, &a[2], a[0], n-2);
                mul_n(odd,    &a[1], a[0], n);

                #pragma unroll
                while (i < n-4) {
                    ++i; mad_row(&even[2*i+2], &odd[2*i], &a[i+1], a[i], n-i-1);
                    ++i; qad_row(&odd[2*i], &even[2*i+2], &a[i+1], a[i], n-i);
                }

                asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
                    : "=r"(even[2*n-4]), "=r"(even[2*n-3])
                    : "r"(a[n-1]), "r"(a[n-3]));
                asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
                    : "+r"(odd[2*n-6]), "+r"(odd[2*n-5])
                    : "r"(a[n-2]), "r"(a[n-3]));
                asm("addc.u32 %0, %0, 0;" : "+r"(even[2*n-3]));

                asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
                    : "=r"(odd[2*n-4]), "=r"(odd[2*n-3])
                    : "r"(a[n-1]), "r"(a[n-2]));

                // merge |even[2:]| and |odd[1:]|
                cadd_n(&even[2], &odd[1], 2*n-4);
                asm("addc.u32 %0, %1, 0;" : "=r"(even[2*n-2]) : "r"(odd[2*n-3]));

                // double |even|
                even[0] = 0;
                asm("add.cc.u32 %0, %1, %1;" : "=r"(even[1]) : "r"(odd[0]));
                for (j = 2; j < 2*n-1; j++)
                    asm("addc.cc.u32 %0, %0, %0;" : "+r"(even[j]));
                asm("addc.u32 %0, 0, 0;" : "=r"(even[j]));

                // accumulate "diagonal" |a[i]|*|a[i]| product
                i = 0;
                asm("mad.lo.cc.u32 %0, %2, %2, %0; madc.hi.cc.u32 %1, %2, %2, %1;"
                    : "+r"(even[2*i]), "+r"(even[2*i+1])
                    : "r"(a[i]));
                for (++i; i < n; i++)
                    asm("madc.lo.cc.u32 %0, %2, %2, %0; madc.hi.cc.u32 %1, %2, %2, %1;"
                        : "+r"(even[2*i]), "+r"(even[2*i+1])
                        : "r"(a[i]));
            }
        };

    private:
        inline operator const uint32_t*() const             { return even;    }
        inline operator uint32_t*()                         { return even;    }

    public:
        inline uint32_t& operator[](size_t i)               { return even[i]; }
        inline const uint32_t& operator[](size_t i) const   { return even[i]; }
        inline size_t len() const                           { return n;       }

        inline mont_t() {}
        // write in 
        inline mont_t(const uint32_t *p)
        {
            for (size_t i = 0; i < n; i++)
                even[i] = p[i];
        }


    // write back
        inline void store(uint32_t *p) const
        {
            for (size_t i = 0; i < n; i++)
                p[i] = even[i];
        }

        inline mont_t& operator+=(const mont_t& b)
        {
            cadd_n(&even[0], &b[0]);
            final_subc();
            return *this;
        }
        friend inline mont_t operator+(mont_t a, const mont_t& b)
        {   return a += b;   }

        friend inline mont_t operator<<(mont_t a, unsigned l)
        {   return a <<= l;   }

        inline mont_t& operator>>=(unsigned r)
        {
            size_t i;
            uint32_t tmp[n+1];

            while (r--) {
                tmp[n] = 0 - (even[0]&1);
                for (i = 0; i < n; i++)
                    tmp[i] = MOD[i] & tmp[n];

                cadd_n(&tmp[0], &even[0]);
                if (N%32 == 0)
                    asm("addc.u32 %0, 0, 0;" : "=r"(tmp[n]));

                for (i = 0; i < n-1; i++)
                    asm("shf.r.wrap.b32 %0, %1, %2, 1;"
                        : "=r"(even[i]) : "r"(tmp[i]), "r"(tmp[i+1]));
                if (N%32 == 0)
                    asm("shf.r.wrap.b32 %0, %1, %2, 1;"
                        : "=r"(even[i]) : "r"(tmp[i]), "r"(tmp[i+1]));
                else
                    even[i] = tmp[i] >> 1;
            }

            return *this;
        }
        friend inline mont_t operator>>(mont_t a, unsigned r)
        {   return a >>= r;   }

        inline mont_t& operator-=(const mont_t& b)
        {
            size_t i;
            uint32_t tmp[n], borrow;

            asm("sub.cc.u32 %0, %0, %1;" : "+r"(even[0]) : "r"(b[0]));
            for (i = 1; i < n; i++)
                asm("subc.cc.u32 %0, %0, %1;" : "+r"(even[i]) : "r"(b[i]));
            asm("subc.u32 %0, 0, 0;" : "=r"(borrow));

            asm("add.cc.u32 %0, %1, %2;" : "=r"(tmp[0]) : "r"(even[0]), "r"(MOD[0]));
            for (i = 1; i < n-1; i++)
                asm("addc.cc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));
            asm("addc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));

            asm("{ .reg.pred %top; setp.ne.u32 %top, %0, 0;" :: "r"(borrow));
            for (i = 0; i < n; i++)
                asm("@%top mov.b32 %0, %1;" : "+r"(even[i]) : "r"(tmp[i]));
            asm("}");

            return *this;
        }
        friend inline mont_t operator-(mont_t a, const mont_t& b)
        {   return a -= b;   }

        inline mont_t operator-() const
        {   return cneg(*this, true);   }

    private:
        static inline void madc_n_rshift(uint32_t* odd, const uint32_t *a, uint32_t bi)
        {
            for (size_t j = 0; j < n-2; j += 2)
                asm("madc.lo.cc.u32 %0, %2, %3, %4; madc.hi.cc.u32 %1, %2, %3, %5;"
                    : "=r"(odd[j]), "=r"(odd[j+1])
                    : "r"(a[j]), "r"(bi), "r"(odd[j+2]), "r"(odd[j+3]));
            asm("madc.lo.cc.u32 %0, %2, %3, 0; madc.hi.u32 %1, %2, %3, 0;"
                : "=r"(odd[n-2]), "=r"(odd[n-1])
                : "r"(a[n-2]), "r"(bi));
        }

        static inline void mad_n_redc(uint32_t *even, uint32_t* odd,
                                    const uint32_t *a, uint32_t bi, bool first=false)
        {
            if (first) {
                mul_n(odd, a+1, bi);
                mul_n(even, a,  bi);
            } else {
                asm("add.cc.u32 %0, %0, %1;" : "+r"(even[0]) : "r"(odd[1]));
                madc_n_rshift(odd, a+1, bi);
                cmad_n(even, a, bi);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            }

            uint32_t mi = even[0] * M0;

            cmad_n(odd, MOD+1, mi);
            cmad_n(even, MOD,  mi);
            asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
        }

    public:
        friend inline mont_t operator*(const mont_t& a, const mont_t& b)
        {
            if (N%32 == 0) {
                return wide_t{a, b};
            } else {
                mont_t even, odd;

                #pragma unroll
                for (size_t i = 0; i < n; i += 2) {
                    mad_n_redc(&even[0], &odd[0], &a[0], b[i], i==0);
                    mad_n_redc(&odd[0], &even[0], &a[0], b[i+1]);
                }

                // merge |even| and |odd|
                cadd_n(&even[0], &odd[1], n-1);
                asm("addc.u32 %0, %0, 0;" : "+r"(even[n-1]));

                even.final_sub(0, &odd[0]);

                return even;
            }
        }
        inline mont_t& operator*=(const mont_t& a)
        {   return *this = *this * a;   }

        inline mont_t& sqr()
        {   return *this = wide_t{*this};   }

        inline mont_t operator()(uint32_t p)
        {   return *this^p;   }

        // raise to a constant power, e.g. x^7, to be unrolled at compile time
        inline mont_t& operator^=(int p)
        {
            if (p < 2)
                asm("trap;");

            mont_t sqr = *this;
            if ((p&1) == 0) {
                do {
                    sqr.sqr();
                    p >>= 1;
                } while ((p&1) == 0);
                *this = sqr;
            }
            for (p >>= 1; p; p >>= 1) {
                sqr.sqr();
                if (p&1)
                    *this *= sqr;
            }
            return *this;
        }
        friend inline mont_t operator^(mont_t a, int p)
        {   return p == 2 ? (mont_t)wide_t{a} : a ^= p;   }
        inline mont_t operator()(int p)
        {   return *this^p;   }
        friend inline mont_t sqr(const mont_t& a)
        {   return a^2;   }

        inline void to()    { mont_t t = RR * *this; *this = t; }
        inline void to(const uint32_t a[2*n], bool host_order = true)
        {
            size_t i;

            // load the most significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    even[i] = a[n + i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, %1, 0x0123;" : "=r"(even[i]) : "r"(a[n - 1 - i]));
            }
            to();

            mont_t lo;

            // load the least significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    lo[i] = a[i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, %1, 0x0123;" : "=r"(lo[i]) : "r"(a[2*n - 1 - i]));
            }

            cadd_n(&even[0], &lo[0]);
            final_subc();
            to();
        }
        inline void from()  { mont_t t = *this; t.mul_by_1(); *this = t; }
        inline void from(const uint32_t a[2*n], bool host_order = true)
        {
            size_t i;

            // load the least significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    even[i] = a[i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, 0, 0x0123;" : "=r"(even[i]) : "r"(a[2*n - 1 -i]));
            }
            mul_by_1();

            mont_t hi;

            // load the most significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    hi[i] = a[n + i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, 0, 0x0123;" : "=r"(hi[i]) : "r"(a[n - 1 - i]));
            }

            cadd_n(&even[0], &hi[0]);
            final_subc();
            to();
        }

    public:
        static inline void mul_by_1_row(uint32_t* even, uint32_t* odd, bool first=false)
        {
            uint32_t mi;

            if (first) {
                mi = even[0] * M0;
                mul_n(odd, MOD+1, mi);
                cmad_n(even, MOD,  mi);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            } else {
                asm("add.cc.u32 %0, %0, %1;" : "+r"(even[0]) : "r"(odd[1]));
    # if 1      // do we trust the compiler to *not* touch the carry flag here?
                mi = even[0] * M0;
    # else
                asm("mul.lo.u32 %0, %1, %2;" : "=r"(mi) : "r"(even[0]), "r"(M0));
    # endif
                madc_n_rshift(odd, MOD+1, mi);
                cmad_n(even, MOD, mi);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            }
        }
        inline void mul_by_1()
        {
            mont_t odd;

            #pragma unroll
            for (size_t i = 0; i < n; i += 2) {
                mul_by_1_row(&even[0], &odd[0], i==0);
                mul_by_1_row(&odd[0], &even[0]);
            }

            cadd_n(&even[0], &odd[1], n-1);
            asm("addc.u32 %0, %0, 0;" : "+r"(even[n-1]));
        }

        inline void final_sub(uint32_t carry, uint32_t* tmp)
        {
            size_t i;
            asm("{ .reg.pred %top;");

            asm("sub.cc.u32 %0, %1, %2;" : "=r"(tmp[0]) : "r"(even[0]), "r"(MOD[0]));
            for (i = 1; i < n; i++)
                asm("subc.cc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));
            if (N%32 == 0)
                asm("subc.u32 %0, %0, 0; setp.eq.u32 %top, %0, 0;" : "+r"(carry));
            else
                asm("subc.u32 %0, 0, 0; setp.eq.u32 %top, %0, 0;" : "=r"(carry));

            for (i = 0; i < n; i++)
                asm("@%top mov.b32 %0, %1;" : "+r"(even[i]) : "r"(tmp[i]));

            asm("}");
        }

        inline void final_subc()
        {
            uint32_t carry, tmp[n];

            asm("addc.u32 %0, 0, 0;" : "=r"(carry));

            asm("sub.cc.u32 %0, %1, %2;" : "=r"(tmp[0]) : "r"(even[0]), "r"(MOD[0]));
            for (size_t i = 1; i < n; i++)
                asm("subc.cc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));
            asm("subc.u32 %0, %0, 0;" : "+r"(carry));

            asm("{ .reg.pred %top;");
            asm("setp.eq.u32 %top, %0, 0;" :: "r"(carry));
            for (size_t i = 0; i < n; i++)
                asm("@%top mov.b32 %0, %1;" : "+r"(even[i]) : "r"(tmp[i]));
            asm("}");
        }

};


typedef mont_t<254, device::ALT_BN128_P, device::ALT_BN128_M0,
                    device::ALT_BN128_RR, device::ALT_BN128_one,
                    device::ALT_BN128_Px4> fp_mont;


typedef mont_t<753, device::MNT4753_Fr_P, device::MNT4753_Fr_M0,
                    device::MNT4753_Fr_P, device::MNT4753_Fr_P,
                    device::MNT4753_Fr_P> MNT4753_Fr;

typedef mont_t<1024, device::Curve1024_P0, device::Curve1024_M0,
                    device::Curve1024_P0, device::Curve1024_P0,
                    device::Curve1024_P0> CURVE1024_Fr;

typedef mont_t<2047, device::Curve2048_P0, device::Curve2048_M0,
                    device::Curve2048_P0, device::Curve2048_P0,
                    device::Curve2048_P0> CURVE2048_Fr;

typedef mont_t<4095, device::Curve4096_P0, device::Curve4096_M0,
                    device::Curve4096_P0, device::Curve4096_P0,
                    device::Curve4096_P0> CURVE4096_Fr;






void __global__ __launch_bounds__(1024)  func1(uint32_t res[]){
    const uint32_t* p;
    const uint32_t* p1;
    uint32_t arr[128] =  {0x9c0a6f42
        ,0xebb6a0ba
        ,0x37dd94ef
        ,0x6d79fb73
        ,0x7983207e
        ,0x0b9e00b7
        ,0xd8fa3981
        ,0xbf6b3ed2
        ,0xec47b3b6
        ,0x02b47949
        ,0xdcc7abf4
        ,0x0f18044b
        ,0x0906e354
        ,0x672cfe00
        ,0x65a9f075
        ,0xd0caa5f4
        ,0x9a73eaff
        ,0x87604c0d
        ,0x740a5b97
        ,0xb424b881
        ,0xa415e28c
        ,0x2b0009ce
        ,0x95ba33c9
        ,0x25671aba
        ,0x68a56e13
        ,0xae44df47
        ,0x6d6ba5f2
        ,0x08bfd4fe
        ,0xa3697a7e
        ,0x62509cd1
        ,0xc81350fb
        ,0x80b6959c
        ,0xcead05e5
        ,0xd246923c
        ,0xb8a36df5
        ,0xfc512028
        ,0x80dc54fa
        ,0xca57a9b0
        ,0xd504f7bd
        ,0x23596578
        ,0x635d3b55
        ,0x0bb6b26e
        ,0x1676d5bb
        ,0xcb6a7548
        ,0x8dc10ee7
        ,0x5b7c2a1f
        ,0xff779b41
        ,0x3b2848ab
        ,0xcc55cc16
        ,0xcefa60c1
        ,0xd65c561c
        ,0xbb70f64d
        ,0x16d831ba
        ,0x3d436af7
        ,0x77f44368
        ,0xd5f011cc
        ,0x48cdef86
        ,0x7aafa9c6
        ,0x107e0360
        ,0x9c362b0c
        ,0xba8f2d7d
        ,0xc087ce45
        ,0x802c7068
        ,0x7981ddd6
        ,0x05a91b19
        ,0xcadd0630
        ,0xf19bbfdb
        ,0xdf78d97e
        ,0x1ce8d6a8
        ,0x73a8d43e
        ,0xeb02456d
        ,0x5011b85d
        ,0x371112b9
        ,0x8e7d207b
        ,0x433e21ff
        ,0xe48a2325
        ,0xa322cd23
        ,0xd4ac2f85
        ,0x0dbc3dc9
        ,0xcb2d7570
        ,0x1ffc433a
        ,0x7f647e33
        ,0x8fe481c9
        ,0x88cf8dba
        ,0x28f3ec37
        ,0x2a42faf9
        ,0xe891add9
        ,0x7f861ba8
        ,0x4e2ffe49
        ,0xf920f7d1
        ,0x2597f4c6
        ,0x3dc82277
        ,0x868f4226
        ,0xe732b14b
        ,0x5de21876
        ,0xa0f5e592
        ,0x45dbe2ca
        ,0xb566d1a5
        ,0xa705ead2
        ,0x69beaf1c
        ,0x90504600
        ,0xdfd5be24
        ,0xff88d1e9
        ,0xd460c1e8
        ,0x439e2934
        ,0xd97b88e2
        ,0x2de2d2c5
        ,0x746cc9a6
        ,0x8fedd66d
        ,0xe70e2a4b
        ,0x731c389b
        ,0xb44c21b9
        ,0xe48dcf39
        ,0xd73aae49
        ,0x0d585594
        ,0xf8e90c92
        ,0x4ef6861a
        ,0x752c7c90
        ,0x5ba3a788
        ,0xe078f1a0
        ,0xcb291037
        ,0x07309e0c
        ,0xa8cb7709
        ,0x40642ff8
        ,0x3c779f00
        ,0xaaaeee78
        ,0xf0ed822d
        ,0xc0b19ee5};

    // uint32_t arr[8] =  {0xd87cfd47,0x3c208c16,0x6871ca8d,0x97816a91,0x8181585d,0xb85045b6,0xe131a029,0x30644e72};//p,module
    // uint32_t arr[8] =  {0x11111111,0x11111111,0x11111111,0x11111111,0x11111111,0x11111111,0x11111111,0x11111111};
    p = arr;
    CURVE4096_Fr exampleInstance(p);


    // uint32_t arr1[8] = {0x0,0x0,0x0,0x0,0x0,0x0,0x0,0x1};
    // R
        // TO_CUDA_T(0xd35d438dc58f0d9d), TO_CUDA_T(0x0a78eb28f5c70b3a),
        // TO_CUDA_T(0x666ea36f7879462c), TO_CUDA_T(0x0e0a77c19a07df2f)
    // uint32_t arr1[8] ={0x538afa89,0xf32cfc5b,0xd44501fb,0xb5e71911,0x0a417ff6,0x47ab1eff,0xcab8351f,0x06d89f71};//RR
    uint32_t arr1[128] ={0x5c285796
        ,0x8d149319
        ,0x490a1357
        ,0x52db1454
        ,0x3939def3
        ,0xa251ea68
        ,0xa692d350
        ,0x355ef91e
        ,0xf3e941d8
        ,0xe6a1c9a1
        ,0xfd199199
        ,0x6af6cf58
        ,0xda500a44
        ,0x666ecd51
        ,0x30e94bc3
        ,0xcd87ca7b
        ,0x314f8097
        ,0xc01a4b6f
        ,0xcb2ff23e
        ,0x502bd0c8
        ,0x6f0ed927
        ,0xb4a6e691
        ,0x0053b87c
        ,0xd73d030b
        ,0x0c6c3f49
        ,0x025c3a54
        ,0x1b84ed45
        ,0xe1810db6
        ,0x47033d9b
        ,0xb8a18bb4
        ,0xb5dcfe75
        ,0x8b9b3738
        ,0x0845dc46
        ,0xea5795c7
        ,0x6b2c9e60
        ,0x70b3b87f
        ,0xc11d49be
        ,0x32211c2f
        ,0xea06a6be
        ,0x03416da9
        ,0xbe4d2f01
        ,0x556e7ba8
        ,0x79d15e48
        ,0xc4f58684
        ,0xb288ba55
        ,0x63ec61dd
        ,0xa1d1d5d5
        ,0x2bd4de93
        ,0x91985701
        ,0xa5585098
        ,0x5358cf43
        ,0x3d5e8964
        ,0xeb06da5c
        ,0x39c77210
        ,0x71017c47
        ,0x0aa1929e
        ,0xbb07f827
        ,0x97707483
        ,0x30633109
        ,0x1bd60690
        ,0x0086fa8f
        ,0x97ec267b
        ,0xc2e4ba81
        ,0x970ddddf
        ,0x91ec010f
        ,0xd9da115f
        ,0xdcdd5ebb
        ,0x10bcb89e
        ,0xbcf79830
        ,0xde664755
        ,0x13f0c4e6
        ,0x027940bc
        ,0x999b7504
        ,0xe3c0b470
        ,0x35b16d65
        ,0xa762d0ae
        ,0xce06ea6b
        ,0xa83a6bde
        ,0x7832d36e
        ,0x13eff3af
        ,0xa4d7760d
        ,0x2f2e226e
        ,0xcbcdd58a
        ,0x4be5c82f
        ,0x42039e4b
        ,0x5c946e19
        ,0xe867a5a0
        ,0x3821cabc
        ,0xa911702d
        ,0x1a6da9e7
        ,0xa6fafdac
        ,0x533eef0a
        ,0x2baa583b
        ,0x600aaa64
        ,0x8a068ee5
        ,0xcc4462a0
        ,0xaa7413fc
        ,0xf80f5f25
        ,0x54020445
        ,0x557170b7
        ,0xad64a4fe
        ,0x6560445a
        ,0x4426fa51
        ,0x75b8768f
        ,0xa36ffaf6
        ,0x2dd20150
        ,0xb729d555
        ,0xc1e5f401
        ,0xd6226ee2
        ,0x10390572
        ,0x1f342167
        ,0x4a857f94
        ,0xd0d260df
        ,0x04762b8a
        ,0xd469af7d
        ,0xaa83490f
        ,0x3961b48d
        ,0x23f9eae6
        ,0xf0b7811d
        ,0x6fed5763
        ,0x6efe2512
        ,0x378f015b
        ,0x492144e3
        ,0x88448a18
        ,0x284374c6
        ,0xa06f5a0c
        ,0x277800b2
        ,0xdfeebc2f};
    p1 = arr1;
    CURVE4096_Fr exampleInstance1(p1);
    exampleInstance*=exampleInstance1;
    for(int i=0;i<exampleInstance.n;i++){
        res[i]=exampleInstance[i];
    }
    
     return;
}

# undef inline
# undef asm
#endif



#include<iostream>
int main(){
    uint32_t res[128]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    uint32_t *res_d;
    hipMalloc((void**)&res_d,128*sizeof(uint32_t));
    hipMemcpy(res_d, res,128*sizeof(uint32_t), hipMemcpyHostToDevice);



int GRIDEDIM = 1;
while (true)
{
    printf("gridDIM = %d\n",GRIDEDIM);


int NUM_REPEATS = 10;
 float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipEventQuery(start);

    func1<<<GRIDEDIM,1024>>>(res_d);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    
    printf("Time = %g +- %g ms.\n", t_ave, t_err);



GRIDEDIM=GRIDEDIM*10;
if(GRIDEDIM==10000000){
    break;
}
}



    
    hipMemcpy(res,res_d,128*sizeof(uint32_t),hipMemcpyDeviceToHost);
    for(int i=0;i<128;i++){
        std::cout<<std::hex<<res[i]<<std::endl;
    }
    return 0;
}